#include "hip/hip_runtime.h"
#include "NumTest_gpu.hpp"

namespace NumTest_gpu
{
    __global__ void test_dot(double *c, const double *a, const double *b, const int WIDTH)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int i = y * WIDTH + x;

        double sum = 0.0f;
        for (int k = 0; k < WIDTH; ++k)
        {
            double lhs = a[y * WIDTH + k];
            double rhs = b[k * WIDTH + x];
            sum += lhs * rhs;
        }
        c[i] = sum;
    }
    
    void test_dot_gpu()
    {
        int WIDTH = 512;
        int TILE_WIDTH = 16;
        int GRID_WIDTH = WIDTH / TILE_WIDTH;

        double a[WIDTH][WIDTH];
        double b[WIDTH][WIDTH];
        double c[WIDTH][WIDTH] = {0};

        for (int y = 0; y < WIDTH; ++y)
        {
            for (int x = 0; x < WIDTH; ++x)
            {
                a[y][x] = 1.0;
                b[y][x] = 1.0;
            }
        }

        double *dev_a = 0;
        double *dev_b = 0;
        double *dev_c = 0;
        CUDA_CHECK(hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(double)));
        CUDA_CHECK(hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(double)));
        CUDA_CHECK(hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(double)));

        CUDA_CHECK(hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(double), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(double), hipMemcpyHostToDevice));

        dim3 dimGrid(GRID_WIDTH, GRID_WIDTH, 1);
        dim3 dimThread(TILE_WIDTH, TILE_WIDTH, 1);
        for (int i = 0; i < 128; ++i)
            test_dot<<<dimGrid, dimThread>>>(dev_c, dev_a, dev_b, WIDTH);

        CUDA_CHECK(hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(double), hipMemcpyDeviceToHost));
        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);

        for (int y = 0; y < WIDTH; ++y)
        {
            for (int x = 0; x < WIDTH; ++x)
            {
                printf("%lf ", c[y][x]);
            }
            printf("\n");
        }
    }

    void copy_gpu_to_gpu(size_t size, double* out_dev_data, const double* in_dev_data)
    {
        CUDA_CHECK(hipMemcpy(out_dev_data, in_dev_data, size, hipMemcpyDeviceToDevice));
    }

    void copy_cpu_to_gpu(size_t size, double* dev_data, const double* data)
    {
        CUDA_CHECK(hipMemcpy(dev_data, data, size, hipMemcpyHostToDevice));
    }

    void copy_gpu_to_cpu(size_t size, double* data, const double* dev_data)
    {
        CUDA_CHECK(hipMemcpy(data, dev_data, size, hipMemcpyDeviceToHost));
    }

    double* gpu_malloc(size_t size)
    {
        double* dev_data=nullptr;
        CUDA_CHECK(hipMalloc((void**)&dev_data, size));
        return dev_data;
    }

    void* cpu_malloc(size_t size)
    {
        return malloc(size);
    }

    void gpu_free(double* dev_data)
    {
        if (dev_data)
        {
            CUDA_CHECK(hipFree(dev_data));
        }
    }

    void cpu_free(void* data)
    {
        if (data)
        {
            free(data);
        }
    }

    __global__ void transpose(double* out_dev_data, const double* in_dev_data, const size_t in_rows, const size_t in_cols)
    {
        size_t x = threadIdx.x;
        size_t y = threadIdx.y;

        out_dev_data[x * in_rows + y] = in_dev_data[y * in_cols + x];
    }

    void transpose_gpu(double* out_dev_data, const double* in_dev_data, const size_t in_rows, const size_t in_cols)
    {
        dim3 dimGrid(1, 1, 1);
        dim3 dimThread(in_cols, in_rows, 1);

        transpose<<<dimGrid, dimThread>>>(out_dev_data, in_dev_data, in_rows, in_cols);
    }

    __global__ void matrix_dot(double *dev_out, const double* dev_lhs, const double* dev_rhs, 
        const size_t lhs_rows, const size_t lhs_cols, const size_t rhs_rows, const size_t rhs_cols)
    {
        size_t x = threadIdx.x;
        size_t y = threadIdx.y;
        size_t i = y * rhs_cols + x;

        double sum = 0.0;

        for (size_t k = 0; k < lhs_cols; ++k)
        {
            sum += dev_lhs[y * lhs_cols + k] * dev_rhs[k * rhs_cols + x];
        }

        dev_out[i] = sum;
    }

    double* matrix_dot_gpu(double* dev_out, const double* dev_lhs, const double* dev_rhs, 
        const size_t lhs_rows, const size_t lhs_cols, const size_t rhs_rows, const size_t rhs_cols)
    {
        dim3 dimGrid(1, 1, 1);
        dim3 dimThread(rhs_cols, lhs_rows, 1);

        matrix_dot<<<dimGrid, dimThread>>>(dev_out, dev_lhs, dev_rhs, lhs_rows, lhs_cols, rhs_rows, rhs_cols);
        return dev_out;
    }

    __global__ void add(double* dev_out, const double* dev_lhs, const double* dev_rhs)
    {
        size_t i = threadIdx.y * blockDim.x + threadIdx.x;
        dev_out[i] = dev_lhs[i] + dev_rhs[i];
    }

    double* add_gpu(double* dev_out, const double* dev_lhs, const double* dev_rhs, const size_t& rows, const size_t& cols)
    {
        dim3 dimGrid(1, 1, 1);
        dim3 dimBlock(cols, rows, 1);

        add<<<dimGrid, dimBlock>>>(dev_out, dev_lhs, dev_rhs);
        return dev_out;
    }

    __global__ void tanh_(double* out_dev_data, const double* in_dev_data)
    {
        size_t i = threadIdx.y * blockDim.x + threadIdx.x;
        out_dev_data[i] = tanh(in_dev_data[i]);
    }

    void tanh_gpu(double* out_dev_data, const double* in_dev_data, const size_t& rows, const size_t& cols)
    {
        dim3 dimGrid(1, 1, 1);
        dim3 dimBlock(cols, rows, 1);

        tanh_<<<dimGrid, dimBlock>>>(out_dev_data, in_dev_data);
    }
}