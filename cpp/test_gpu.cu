#include "hip/hip_runtime.h"
#include "test_gpu.h"

namespace test_gpu
{
    void test()
    {
        const int SIZE = 5;
        const int a[SIZE] = {1, 2, 3, 4, 5};
        int b[SIZE] = {0, 0, 0, 0, 0};

        printf("a = ");
        for (int i = 0; i < 5; ++i)
        {
            printf("%d ", a[i]);
        }
        printf("\n");

        int *dev_a = 0;
        int *dev_b = 0;

        CUDA_CHECK(hipMalloc((void**)&dev_a, SIZE * sizeof(int)));
        hipMalloc((void**)&dev_b, SIZE * sizeof(int));

        hipMemcpy(dev_a, a, SIZE * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_b, dev_a, SIZE * sizeof(int), hipMemcpyDeviceToDevice);
        hipMemcpy(b, dev_b, SIZE * sizeof(int), hipMemcpyDeviceToHost);


        printf("b = ");
        for (int i = 0; i < 5; ++i)
        {
            printf("%d ", b[i]);
        }
        printf("\n");
    }

    __global__ void addKernel(int *c, const int *a, const int *b)
    {
        int x = threadIdx.x;
        int y = threadIdx.y;

        int i = y * (blockDim.x) + x;
        c[i] = a[i] + b[i];
    }

    void test_matrix_add()
    {
        const int WIDTH = 5;
        int a[WIDTH][WIDTH];
        int b[WIDTH][WIDTH];
        int c[WIDTH][WIDTH] = {0};

        for (int y = 0; y < WIDTH; ++y)
        {
            for (int x = 0; x < WIDTH; ++x)
            {
                a[y][x] = y * 10 + x;
                b[y][x] = a[y][x] * 100;
            }
        }

        int *dev_a = 0;
        int *dev_b = 0;
        int *dev_c = 0;
        CUDA_CHECK(hipMalloc((void**)&dev_a, WIDTH * WIDTH * sizeof(int)));
        CUDA_CHECK(hipMalloc((void**)&dev_b, WIDTH * WIDTH * sizeof(int)));
        CUDA_CHECK(hipMalloc((void**)&dev_c, WIDTH * WIDTH * sizeof(int)));

        CUDA_CHECK(hipMemcpy(dev_a, a, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(dev_b, b, WIDTH * WIDTH * sizeof(int), hipMemcpyHostToDevice));

        dim3 dimGrid(1, 1, 1);
        dim3 dimBlock(WIDTH, WIDTH, 1);
        addKernel<<<dimGrid, dimBlock>>>(dev_c, dev_a, dev_b);    

        CUDA_CHECK(hipMemcpy(c, dev_c, WIDTH * WIDTH * sizeof(int), hipMemcpyDeviceToHost));
        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);

        for (int y = 0; y < WIDTH; ++y)
        {
            for (int x = 0; x < WIDTH; ++x)
            {
                printf("%d ", c[y][x]);
            }
            printf("\n");
        }
    }
}